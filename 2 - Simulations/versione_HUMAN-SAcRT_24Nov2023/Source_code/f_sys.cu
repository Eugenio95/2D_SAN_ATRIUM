#include "hip/hip_runtime.h"
// -*- mode: c++; c-basic-offset: 8; -*-

/*
    F_SYS kernel
    The kernel updates the state variables according to cell types and calculates the currents exchanged by neighbouring cells

*/

#include "koi_2011.cuh"
#include "fab_2017.cuh"
#include "mor_2016.cuh"
#include "update_Vgap.cuh"
#include "f_sys.hpp"
#include "const_def.hpp"

__global__
void f_sys(size_t nCells,       // number of cells
           const double *Ymat,  // nStates * nCells
           double *Y_n_mat,     // nStates * nCells
           size_t Ymat_COLS,    // n. of columns of Ymat
           double *dYmat,       // nStates * nCells
           size_t dYmat_COLS,   // n. of columns of dYmat
           const double *rand_g_mat, // nStates * num_g_rand
           size_t rand_g_COLS,  // n. of columns of rand_g_mat
           const int *V_idx,    // array containing index of membrane voltage state variable in X, different between cell types (size:nCells)
           const double *Cm,    // array containing membrane capacitances, different between cell types (size:nCells)
           const double *gJ,    // matrix containing gap junctional resistance values among cells (size: nCells x 4, every cells has Rgap with its 4 neighbours)
           size_t gJ_COLS,      // n. of columns of gJ
           const int WIDTH,     // width of the 2D tissue matrix (size: scalar)
           const int LENGTH,    // length of the 2D tissue matrix (size: scalar)
           const double time,   // time of simulation at current step
           const double h,      // integration step
           const int *cell_type) // array containing logicals expressing the type of the cell (size: nCells)
{
        const size_t j = threadIdx.x + blockIdx.x * blockDim.x;

        // Threads that are out of bound exit now
        if ( j >= nCells )
                return;

        const double *Y = Ymat + j*Ymat_COLS;
        double *dY = dYmat + j*dYmat_COLS;
        double *Y_n = Y_n_mat + j*Ymat_COLS;
        const double *rand_g = rand_g_mat + j*rand_g_COLS;
        
	for (size_t k = 0; k < Ymat_COLS; k++) {
                dY[k] = 0.0; // [MM] non sono certo che serva...
        }
        switch (cell_type[j]) {
        case idx_atr:
		koi_2011(Y, dY, time, rand_g, j, WIDTH, LENGTH);
                break;
        case idx_san:
		fab_2017(Y, dY, time, rand_g);
                break;
        case idx_fat:
                for (size_t s=0; s<nStates; s++) {
                        dY[s] = Y[s];
                }
                break;
	default: // case idx_fibro:
		mor_2016(Y, dY, time, rand_g);
                break;
                // default:
                // std::cerr << "Unknown cell type: " << cell_type[j] << std::endl;
                // std::abort();
	}
        
	/*if(j == 20000){
		printf("Vm = %f \n", Ymat[Ymat_COLS*j+V_idx[j]]);
	}*/
	
	
	dY[V_idx[j]] += update_Vgap(Ymat,
                                    Ymat_COLS,
                                    V_idx,
                                    Cm[j],
                                    gJ,
                                    gJ_COLS,
                                    j,
                                    WIDTH,
                                    LENGTH); // evaluate Igap

        /*if(j == 20000){
                printf("Vm_post = %f \n\n", Ymat[Ymat_COLS*j+V_idx[j]]);
        }*/


        for (size_t k = 0; k < Ymat_COLS; k++) {
                Y_n[k] = Y[k] + h * dY[k]; // update every temporary state
        }
}
