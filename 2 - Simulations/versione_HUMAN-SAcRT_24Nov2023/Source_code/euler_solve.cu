#include "hip/hip_runtime.h"
// -*- mode: c++; c-basic-offset: 8; -*-

/*
    EULER SOLVE function
    This functions performs numerical integration of the ODEs using the esplicit euler method, cycling in time and on the cells.

*/


#include <math.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <cassert>
#include <algorithm>
#include <iomanip>

#include "const_def.hpp"
#include "euler_solve.hpp"
#include "f_sys.hpp"
#include "hpc.h"

using namespace std;

// n. of threads per block (must be an integer multiple of 32)
#define BLKDIM 640

void euler_solve(vector<vector<double>>& X,            // state vector matrix(size: nCells x nubmer of ODEs per cell)
                 fstream& fV,                          // file to save membrane voltage data of the cells
                 fstream& fT,                          // file to save time vector
                 fstream& fStates,                     // file to save all of the cells' state variables every N steps
                 int WIDTH,                            // width of the 2D tissue matrix (size: scalar)
                 int LENGTH,                           // length of the 2D tissue matrix (size: scalar)
                 int nCells,                           // total number of cells in the tissue matrix (size: scalar)
                 double sim_time,                      // time of the simulation in seconds (size: scalar)
                 const vector<int>& cell_type,         // array containing logicals expressing the type of the cell (size: nCells)
                 const vector<int>& V_idx,             // array containing index of membrane voltage state variable in X, different between cell types (size:nCells)
                 const vector<double>& Cm,             // array containing membrane capacitances, different between cell types (size:nCells)
                 const vector<vector<double>>& gJ,     // matrix containing gap junctional resistance values among cells (size: nCells x 4, every cells has Rgap with its 4 neighbours)
                 const vector<vector<double>>& rand_g, // matrix containing randomized ionic maximal conductances values (size: nCells x 12)
		 const string &output_states_string
		)
{
        vector< vector<double> > X_n(nCells, vector<double>(nStates, 0)); // FIXME: l'ho tolto dai parametri della funzione e dichiarato qui

	    // PARAMETERS
        double t = 0;
        const int sim_steps = (int) (1.2 / integration_step); // sim_time
	int last_2s;
const vector<int>
	        front_cells{23836,
		23837,
		23838,
		23839,
		23840,
		23841,
		23842,
		23843,
		23844,
		23845,
		23956,
		23957,
		23958,
		23959,
		23960,
		23961,
		23962,
		23963,
		23964,
		23965,
		24466,
		24467,
		24468,
		24469,
		24470,
		24471,
		24472,
		24473,
		24474,
		24475,
		24496,
		24497,
		24498,
		24499,
		24500,
		24501,
		24502,
		24503,
		24504,
		24505,
		24526,
		24527,
		24528,
		24529,
		24530,
		24531,
		24532,
		24533,
		24534,
		24535};
        
	/* We need to copy X, X_n, gJ and rand_g into properly-sized
           C-style arrays */
        const size_t X_ROWS = X.size();
        const size_t X_COLS = X[0].size();
        const size_t X_n_ROWS = X_n.size();
        const size_t X_n_COLS = X_n[0].size();
        const size_t gJ_ROWS = gJ.size();
        const size_t gJ_COLS = gJ[0].size();
        const size_t rand_g_ROWS = rand_g.size();
        const size_t rand_g_COLS = rand_g[0].size();

        assert(X_ROWS == (size_t)nCells);
        assert(X_COLS == (size_t)nStates);
        assert(X_n_ROWS == (size_t)nCells);
        assert(X_n_COLS == (size_t)nStates);
        assert(gJ_ROWS == (size_t)nCells);
        assert(gJ_COLS == (size_t)4);
        assert(rand_g_ROWS == (size_t)nCells);
        assert(rand_g_COLS == (size_t)num_g_rand);

        double *X_arr = new double[X_ROWS * X_COLS];
        double *X_n_arr = new double[X_n_ROWS * X_n_COLS];
        double *gJ_arr = new double[gJ_ROWS * gJ_COLS];
        double *rand_g_arr = new double[rand_g_ROWS * rand_g_COLS];
        // Each cell requires an array w1[] of nStates elements.  To
        // simplify porting the code to CUDA, we allocate a matrix
        // with nCells * nStates elements; cell j will operate on row
        // j of that matrix.
        double *w1_arr = new double[nCells * nStates]; assert(w1_arr);
        const int *V_idx_arr = V_idx.data();

        /* Copy data in */
        for (size_t i=0; i<X_ROWS; i++) {
                for (size_t j=0; j<X_COLS; j++) {
                        X_arr[i*X_COLS + j] = X[i][j];
                }
                for (size_t j=0; j<gJ_COLS; j++) {
                        gJ_arr[i*gJ_COLS + j] = gJ[i][j];
                }
                for (size_t j=0; j<rand_g_COLS; j++) {
                        rand_g_arr[i*rand_g_COLS + j] = rand_g[i][j];
                }
        }

        // Definition of device copies of the input vectors
        double *d_X_arr;
        const size_t d_X_arr_SIZE = X_ROWS * X_COLS * sizeof(*d_X_arr);
        double *d_X_n_arr;
        const size_t d_X_n_arr_SIZE = X_n_ROWS * X_n_COLS * sizeof(*d_X_n_arr);
        assert(d_X_arr_SIZE == d_X_n_arr_SIZE);
        double *d_gJ_arr;
        const size_t d_gJ_arr_SIZE = gJ_ROWS * gJ_COLS * sizeof(*d_gJ_arr);
        double *d_rand_g_arr;
        const size_t d_rand_g_arr_SIZE = rand_g_ROWS * rand_g_COLS * sizeof(*d_rand_g_arr);
        double *d_w1_arr;
        const size_t d_w1_arr_SIZE = nCells * nStates * sizeof(*d_w1_arr);
        int *d_V_idx_arr;
        const size_t d_V_idx_arr_SIZE = V_idx.size() * sizeof(*d_V_idx_arr);
        double *d_Cm;
        const size_t d_Cm_SIZE = Cm.size() * sizeof(*d_Cm);
        int *d_cell_type;
        const size_t d_cell_type_SIZE = cell_type.size() * sizeof(*d_cell_type);

        // Data allocation and copy
        cudaSafeCall( hipMalloc( (void**)&d_X_arr, d_X_arr_SIZE ) );
        cudaSafeCall( hipMemcpy(d_X_arr, X_arr, d_X_arr_SIZE, hipMemcpyHostToDevice) );

        cudaSafeCall( hipMalloc( (void**)&d_X_n_arr, d_X_n_arr_SIZE ) );
        // no copy needed

        cudaSafeCall( hipMalloc( (void**)&d_gJ_arr, d_gJ_arr_SIZE ) );
        cudaSafeCall( hipMemcpy(d_gJ_arr, gJ_arr, d_gJ_arr_SIZE, hipMemcpyHostToDevice) );

        cudaSafeCall( hipMalloc( (void**)&d_rand_g_arr, d_rand_g_arr_SIZE ) );
        cudaSafeCall( hipMemcpy(d_rand_g_arr, rand_g_arr, d_rand_g_arr_SIZE, hipMemcpyHostToDevice) );

        cudaSafeCall( hipMalloc( (void**)&d_w1_arr, d_w1_arr_SIZE ) );
        // no copy needed

        cudaSafeCall( hipMalloc( (void**)&d_V_idx_arr, d_V_idx_arr_SIZE ) );
        cudaSafeCall( hipMemcpy(d_V_idx_arr, V_idx_arr, d_V_idx_arr_SIZE, hipMemcpyHostToDevice) );

        cudaSafeCall( hipMalloc( (void**)&d_Cm, d_Cm_SIZE ) );
        cudaSafeCall( hipMemcpy( d_Cm, Cm.data(), d_Cm_SIZE, hipMemcpyHostToDevice) );

        cudaSafeCall( hipMalloc( (void**)&d_cell_type, d_cell_type_SIZE) );
        cudaSafeCall( hipMemcpy( d_cell_type, cell_type.data(), d_cell_type_SIZE, hipMemcpyHostToDevice) );

        const dim3 BLOCK(BLKDIM);
        const dim3 GRID((nCells + BLKDIM - 1)/BLKDIM);

	if (sim_steps > (int) 5/integration_step){
	//	cout << "\n\n" << "!!! OCCHIO !!!" << "\n\n";
		last_2s = (int) (sim_steps - (5/integration_step));
	} else { 
		last_2s = 0;
	}
	cudaCheckError();

        cout << "<< Sim start >>" << endl;

        // Cycle in time
        for(int steps = 0; steps < sim_steps; steps++) { //sim_steps

                f_sys<<< GRID, BLOCK >>>(nCells,
                                         d_X_arr,
                                         d_X_n_arr,
                                         X_COLS,
                                         d_w1_arr,
                                         nStates,
                                         d_rand_g_arr,
                                         rand_g_COLS,
                                         d_V_idx_arr,
                                         d_Cm,
                                         d_gJ_arr,
                                         gJ_COLS,
                                         WIDTH,
                                         LENGTH,
                                         t,
                                         integration_step,
                                         d_cell_type);
//		cout << "Size: " << d_w1_arr_SIZE << endl;
//		cout << "Cols: " << X_COLS << endl;
//		cout << "dCols: " << nStates << endl;
		cudaCheckError();


                //cudaSafeCall( hipMemcpy(d_X_arr, d_X_n_arr, d_X_n_arr_SIZE, hipMemcpyDeviceToDevice) );
                double *tmp = d_X_n_arr;
                d_X_n_arr = d_X_arr;
                d_X_arr = tmp;

                // save voltage vector only every "under_samp" steps
                if (steps > last_2s & steps % under_samp == 0) {
                        cudaSafeCall( hipMemcpy(X_arr, d_X_arr, d_X_arr_SIZE, hipMemcpyDeviceToHost) );
                        for (size_t j=0; j<X_ROWS; j++) {
                                fV << X_arr[j*X_COLS + V_idx_arr[j]] << endl;
                        }
			fT << t << endl; // save time vector

			
			// SAVE ALL STATE VARIABLES FOR CELL AT THE FRONTIERI
			for (size_t j = 0; j < front_cells.size(); j++) {
                         	for (size_t k = 0; k < X_COLS; k++) {
                                       	fStates << X_arr[j*X_COLS + k] << endl;
                                }
	                }
		
		}

		//cout << "step = " << steps << endl;
		//cout << setprecision(15) << X_arr[0*X_COLS+14] << "," << X_arr[19900*X_COLS+14] << "," << X_arr[23840*X_COLS+14] << "," << X_arr[22501+14] << endl;

                // save all states variables of all cells every N steps
                if (steps % 10000 == 0) {
                        cout << "Integrating ODEs... t = " << t << " s" << endl; // print progress
                        
                                /*
                        cudaSafeCall( hipMemcpy(X_arr, d_X_arr, d_X_arr_SIZE, hipMemcpyDeviceToHost) );
			fStates.close();
			fStates.open( output_states_string, ofstream::out | ofstream::trunc );			
			
			for (size_t j = 0; j < X_ROWS; j++) {
                                for (size_t k = 0; k < X_COLS; k++) {
                                        fStates << X_arr[j*X_COLS + k] << endl;
                                }
                        }
                */
                }
                t += integration_step; // advance in time
	}

        /* Copy data out */
        cudaSafeCall( hipMemcpy(X_arr, d_X_arr, d_X_arr_SIZE, hipMemcpyDeviceToHost) );
        cudaSafeCall( hipMemcpy(X_n_arr, d_X_n_arr, d_X_n_arr_SIZE, hipMemcpyDeviceToHost) );
        for (size_t i=0; i<X_ROWS; i++) {
                for (size_t j=0; j<X_COLS; j++) {
                        X[i][j] = X_arr[i*nStates + j];
                        X_n[i][j] = X_n_arr[i*nStates + j];
                }
        }

        delete X_arr;
        delete X_n_arr;
        delete gJ_arr;
        delete rand_g_arr;
        delete w1_arr;

        hipFree( d_X_arr );
        hipFree( d_X_n_arr );
        hipFree( d_gJ_arr );
        hipFree( d_rand_g_arr );
        hipFree( d_w1_arr );
        hipFree( d_V_idx_arr );
        hipFree( d_Cm );
        hipFree( d_cell_type );
}
